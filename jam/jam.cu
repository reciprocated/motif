
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <vector>
#include <string>

__global__
void launch(uint32_t *d_table, char* d_source, uint32_t size, uint8_t *d_lut, uint8_t *d_output) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  while (tidx < size) {
    char c = d_source[tidx];

    if (c == 0x4e) {
      tidx += stride;
      continue;
    }

    int vx = d_table[d_lut[c] - 1];
    int idx = tidx;

    while (true) {
      int wordidx = d_table[5 * vx + 4];

      if (wordidx != 0)
        d_output[wordidx - 1] = 0x31;

      idx += 1;
      if (idx > size || vx == 0)
        break;

      c = d_source[idx];
      if (c == 0x4e || c < 0x41)
        break;

      vx = d_table[5 * vx + d_lut[c] - 1];
    }

    tidx += stride;
  }
}

void match(uint32_t *d_table, char* d_source, uint32_t size, uint8_t *d_lut, uint8_t *d_output, std::vector<uint8_t>& output, std::string& source) {
  hipMemcpy(d_source, source.data(), source.size(), hipMemcpyHostToDevice);
  hipMemcpy(d_output, output.data(), output.size(), hipMemcpyHostToDevice);

  launch<<<8000, 1024>>>(d_table, d_source, source.size(), d_lut, d_output);

  hipMemcpy(output.data(), d_output, output.size(), hipMemcpyDeviceToHost);
}
